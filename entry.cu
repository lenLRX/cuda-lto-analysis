#include "hip/hip_runtime.h"
#include "add_1.cuh"

__global__ void Entry(float* input1, float* input2, float* output1) {
  int tid = threadIdx.x;
  float a = input1[tid];
  float b = input2[tid];
  float c = a + b;
  float d = add_1(c);
  output1[tid] = d;
}

int main() {
  return 0;
}